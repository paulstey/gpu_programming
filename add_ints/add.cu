
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int a, int b, int* c) {
    *c = a + b;
}


int main() {
    int a = 1;
    int b = 3;
    int c;
    int* dev_c;
  
    hipMalloc((void**)&dev_c, sizeof(int));

    add<<<10,1>>>(a, b, dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "c is: " << c << std::endl;

    hipFree(dev_c);
}

