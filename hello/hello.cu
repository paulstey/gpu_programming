
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

const int NUM_BLOCKS = 10;
const int BLOCK_WIDTH = 1024;



__global__ void hello() {
    printf("Hello world! I'm thread %d, in block %d, \n", threadIdx.x, blockIdx.x);
}


int main(int argc,char *argv[]) {
    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    printf("That's all!\n");
}